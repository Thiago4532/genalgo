#include "hip/hip_runtime.h"
#include "CudaFitnessEngine.hpp"

#include <cstdio>
#include <cstdlib>
#include "PoorProfiler.hpp"
#include "Vec.hpp"
#include "defer.hpp"
#include "globalConfig.hpp"

static void cudaCheck(hipError_t error, const char* message) {
    if (error != hipSuccess) {
        std::fprintf(stderr, "CUDA error: %s: %s\n", message, hipGetErrorString(error));
        std::abort();
    }
}

#define CUDA_CHECK(expr) ::cudaCheck(expr, #expr)

GA_NAMESPACE_BEGIN

constexpr i32 THREADS_PER_INDIVIDUAL = 32;

struct VecTriangle {
    Vec2i a, b, c;
    Color color;
};
static_assert(sizeof(VecTriangle) == sizeof(Triangle) &&
              alignof(VecTriangle) == alignof(Triangle), "VecTriangle is not layout-compatible with Triangle");

template<typename T>
inline void deviceMalloc(T** ptr, size_t count) {
    CUDA_CHECK(hipMalloc(ptr, count * sizeof(T)));
}

template<typename T>
inline T* deviceMalloc(size_t count) {
    T* ptr;
    deviceMalloc(&ptr, count);
    return ptr;
}

static __host__ __device__
Vec3f fromColor(Color c) {
    return Vec3f{1.0f * c.r, 1.0f * c.g, 1.0f * c.b};
}

class CudaFitnessEngine::Engine {
public:
    Engine();
    ~Engine();

    void evaluate(std::vector<Individual>& individuals);
private:
    std::vector<VecTriangle> triangles;
    std::vector<f64> fitnesses;
    Vec2i* hostNumTriangles = nullptr;

    f64* deviceFitnesses = nullptr;
    Vec2i* deviceNumTriangles = nullptr;
    Vec3f* deviceCanvas = nullptr;
    Vec3f* deviceImage = nullptr;

    i32 imWidth, imHeight, imSize;
    i64 canvasSize;
    i32 populationSize;
};

CudaFitnessEngine::Engine::Engine() {
    populationSize = globalCfg.populationSize;
    imWidth = globalCfg.targetImage.getWidth();
    imHeight = globalCfg.targetImage.getHeight();
    imSize = imWidth * imHeight;
    canvasSize = static_cast<i64>(imSize) * populationSize;
    if (canvasSize > std::numeric_limits<i32>::max()) {
        // FIXME: Implement support for 64-bits canvas
        std::fprintf(stderr, "CudaFitnessEngine: canvas size is too large\n");
        std::fprintf(stderr, "  This is a limitation of the current implementation, for now :)\n");
        std::abort();
    }

    triangles.reserve(populationSize * 100);
    fitnesses.resize(populationSize);

    hostNumTriangles = new Vec2i[populationSize];

    deviceMalloc(&deviceFitnesses, populationSize);
    deviceMalloc(&deviceCanvas, canvasSize);
    deviceMalloc(&deviceImage, imSize);
    deviceMalloc(&deviceNumTriangles, populationSize);

    Vec3f* hostImage = new Vec3f[imSize];
    defer { delete[] hostImage; };

    Color* target = reinterpret_cast<Color*>(globalCfg.targetImage.getData());
    for (i32 y = 0; y < imHeight; ++y) {
        for (i32 x = 0; x < imWidth; ++x) {
            i32 i = y * imWidth + x;
            hostImage[i] = (target[i].a / 255.0f) * fromColor(target[i]);
        }
    }

    CUDA_CHECK(hipMemcpy(deviceImage, hostImage, imSize * sizeof(Vec3f), hipMemcpyHostToDevice));
}

CudaFitnessEngine::Engine::~Engine() {
    hipFree(deviceFitnesses);
    hipFree(deviceCanvas);
    hipFree(deviceImage);
    hipFree(deviceNumTriangles);

    free(hostNumTriangles);
}

static __device__
bool pointInTriangle(Vec2i p, Vec2i a, Vec2i b, Vec2i c) {
    auto sign = [](Vec2i p1, Vec2i p2, Vec2i p3) {
        return (p1.x - p3.x) * (p2.y - p3.y) - (p2.x - p3.x) * (p1.y - p3.y);
    };

    bool b1, b2, b3;
    b1 = sign(p, a, b) < 0;
    b2 = sign(p, b, c) < 0;
    b3 = sign(p, c, a) < 0;

    return ((b1 == b2) && (b2 == b3));
}

inline static __device__
Vec3f colorBlend(Vec3f dst, Color src) {
    float alpha = src.a / 255.0;

    return dst * (1.0f - alpha) + fromColor(src) * alpha;
}

[[maybe_unused]] static __global__
void drawTriangles(Vec3f* allCanvas, VecTriangle* allTriangles, Vec2i* allNumTriangles, i32 width, i32 height, i32 populationSize) {
    i32 size = width * height;

    i32 xy = blockIdx.x * blockDim.x + threadIdx.x;
    i32 i = blockIdx.y;
    if (xy >= size)
        return;

    Vec3f* canvas = allCanvas + i * size;
    VecTriangle* triangles = allTriangles + allNumTriangles[i].x;
    i32 numTriangles = allNumTriangles[i].y;

    extern __shared__ VecTriangle sharedTriangles[];
    for (i32 j = threadIdx.x; j < numTriangles; j += blockDim.x) {
        sharedTriangles[j] = triangles[j];
    }
    __syncthreads();

    Vec3f pixel = Vec3f{0.0, 0.0, 0.0};

    i32 x = xy % width;
    i32 y = xy / width;

    for (i32 j = 0; j < numTriangles; ++j) {
        VecTriangle const& t = sharedTriangles[j];
        if (pointInTriangle(Vec2i{x, y}, t.a, t.b, t.c)) {
            pixel = colorBlend(pixel, t.color);
        }
    }
    canvas[xy] = pixel;
}

static __global__
void computeFitnessKernel(
        Vec3f* target, Vec3f* canvas, 
        f64* fitness, i32 imWidth, i32 imHeight, i32 populationSize) {
    i32 numTasks = imWidth * imHeight;
    i32 threadId = blockDim.x * blockIdx.x + threadIdx.x;
    i32 i = threadId / THREADS_PER_INDIVIDUAL;
    if (i >= populationSize)
        return;
    i32 j = threadId % THREADS_PER_INDIVIDUAL;

    f64 fitnessSum = 0.0;
    i32 task0 = j * (numTasks / THREADS_PER_INDIVIDUAL) + min(j, numTasks % THREADS_PER_INDIVIDUAL);
    i32 n = numTasks / THREADS_PER_INDIVIDUAL + (j < numTasks % THREADS_PER_INDIVIDUAL);

    i32 imSize = imWidth * imHeight;
    for (i32 xy = task0; xy < task0 + n; ++xy) {
        Vec3f imgPixel = target[xy];
        Vec3f pixel = canvas[i * imSize + xy];

        double dx = imgPixel.x - pixel.x;
        double dy = imgPixel.y - pixel.y;
        double dz = imgPixel.z - pixel.z;

        fitnessSum += dx * dx + dy * dy + dz * dz;   
    }

    atomicAdd(&fitness[i], fitnessSum);
}

void CudaFitnessEngine::Engine::evaluate(std::vector<Individual>& individuals) {
    if (individuals.size() != populationSize) {
        std::fprintf(stderr, "CudaFitnessEngine::evaluate: individuals.size() != populationSize\n");
        std::abort();
    }

    defer { profiler.stop("cudaFitness:cleanup"); };

    std::vector<VecTriangle> hostTriangles(triangles.size());

    i32 maxTriangles = 0;
    triangles.clear();
    for (i32 i = 0; i < populationSize; ++i) {
        Individual const& ind = individuals[i];
        hostNumTriangles[i].x = triangles.size();
        hostNumTriangles[i].y = ind.size();
        maxTriangles = std::max(maxTriangles, ind.size());
        for (Triangle const& t : ind) {
            VecTriangle vt;
            vt.a = Vec2i(t.a.x, t.a.y);
            vt.b = Vec2i(t.b.x, t.b.y);
            vt.c = Vec2i(t.c.x, t.c.y);
            vt.color = t.color;
            triangles.push_back(vt);
        }
    }

    profiler.start("cudaFitness:copy2device", "Copy");
    auto deviceTriangles = deviceMalloc<VecTriangle>(triangles.size());
    defer { hipFree(deviceTriangles); };
    hipMemcpy(deviceTriangles, triangles.data(), triangles.size() * sizeof(VecTriangle), hipMemcpyHostToDevice);
    hipMemcpy(deviceNumTriangles, hostNumTriangles, populationSize * sizeof(u32), hipMemcpyHostToDevice);
    hipDeviceSynchronize(); // FIXME: Remove this
    profiler.stop("cudaFitness:copy2device");

    profiler.start("cudaFitness:draw", "Draw");
    {
        u32 THREADS = 256;

        dim3 BLOCKS;
        BLOCKS.x = (imSize + THREADS - 1) / THREADS;
        BLOCKS.y = populationSize;
        BLOCKS.z = 1;

        drawTriangles<<<BLOCKS, THREADS,
            maxTriangles * sizeof(VecTriangle)
        >>>(deviceCanvas, deviceTriangles, deviceNumTriangles,
                imWidth, imHeight, populationSize);

        // u32 BLOCKS = (imSize + THREADS - 1) / THREADS;
        // i32 numTriangles = 0;
        // for (i32 i = 0; i < populationSize; ++i) {
        //     drawTriangles<<<BLOCKS, THREADS>>>(deviceCanvas + i * imSize,
        //             deviceTriangles + numTriangles, individuals[i].size(),
        //             imWidth, imHeight, populationSize);
        //     numTriangles += individuals[i].size();
        // }
        hipDeviceSynchronize(); // FIXME: Remove this
    }
    profiler.stop("cudaFitness:draw");

    profiler.start("cudaFitness:compute", "Compute");
    i32 N = populationSize * THREADS_PER_INDIVIDUAL;
    u32 THREADS = 128;
    u32 BLOCKS = (N + THREADS - 1) / THREADS;
    hipMemset(deviceFitnesses, 0, populationSize * sizeof(f64));
    computeFitnessKernel<<<BLOCKS, THREADS>>>(
            deviceImage, deviceCanvas, deviceFitnesses, imWidth, imHeight, populationSize);
    CUDA_CHECK(hipMemcpy(fitnesses.data(), deviceFitnesses, populationSize * sizeof(f64), hipMemcpyDeviceToHost));
    hipDeviceSynchronize();
    profiler.stop("cudaFitness:compute");

    profiler.start("cudaFitness:copy2individuals", "Copy to individuals");
    for (i32 i = 0; i < populationSize; ++i) {
        individuals[i].setFitness(fitnesses[i]);
    }
    profiler.stop("cudaFitness:copy2individuals");


    profiler.start("cudaFitness:cleanup", "Cleanup");
}

// Wrapper for the actual implementation of the engine

CudaFitnessEngine::CudaFitnessEngine() :
    impl(std::make_unique<Engine>()) {}

CudaFitnessEngine::~CudaFitnessEngine() = default;

void CudaFitnessEngine::evaluate(std::vector<Individual>& individuals) {
    impl->evaluate(individuals);
}

GA_NAMESPACE_END

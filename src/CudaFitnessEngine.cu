#include "hip/hip_runtime.h"
#include "CudaFitnessEngine.hpp"

#include <cstdio>
#include <cstdlib>
#include <unistd.h>
#include "PoorProfiler.hpp"
#include "Vec.hpp"
#include "defer.hpp"
#include "GlobalConfig.hpp"

static void cudaCheck(hipError_t error, const char* message) {
    if (error != hipSuccess) {
        std::fprintf(stderr, "CUDA error: %s: %s\n", message, hipGetErrorString(error));
        std::abort();
    }
}

#define CUDA_CHECK(expr) ::cudaCheck(expr, #expr)

GA_NAMESPACE_BEGIN

constexpr i32 THREADS_PER_INDIVIDUAL = 32;

struct VecTriangle {
    Vec2i a, b, c;
    Color color;
};
static_assert(sizeof(VecTriangle) == sizeof(Triangle) &&
        alignof(VecTriangle) == alignof(Triangle), "VecTriangle is not layout-compatible with Triangle");

struct OptimizedVecTriangle {
    Vec2i a, b, c;
    Vec3f color;
    f32 alpha;
    bool use;
};

struct IndividualInfo {
    i32 offset;
    i32 size;
};

template<typename T>
inline void deviceMalloc(T** ptr, size_t count) {
    CUDA_CHECK(hipMalloc(ptr, count * sizeof(T)));
}

template<typename T>
inline T* deviceMalloc(size_t count) {
    T* ptr;
    deviceMalloc(&ptr, count);
    return ptr;
}

template<typename T>
inline void copyHostToDevice(T* device, T const* host, size_t count) {
    CUDA_CHECK(hipMemcpy(device, host, count * sizeof(T), hipMemcpyHostToDevice));
}

template<typename T>
inline void copyDeviceToHost(T* host, T const* device, size_t count) {
    CUDA_CHECK(hipMemcpy(host, device, count * sizeof(T), hipMemcpyDeviceToHost));
}

inline static __host__ __device__
Vec3f fromColor(Color c) {
    return Vec3f{1.0f * c.r, 1.0f * c.g, 1.0f * c.b};
}

class CudaFitnessEngine::Engine {
public:
    Engine();
    ~Engine();

    void evaluate(std::vector<Individual>& individuals);
private:
    std::vector<VecTriangle> triangles;
    std::vector<f64> fitnesses;
    IndividualInfo* hostIndividualInfo = nullptr;

    f64* deviceFitnesses = nullptr;
    IndividualInfo* deviceIndividualInfo = nullptr;
    Vec3f* deviceCanvas = nullptr;
    Vec3f* deviceImage = nullptr;

    i32 imWidth, imHeight, imSize;
    i64 canvasSize;
    i32 populationSize;
};

CudaFitnessEngine::Engine::Engine() {
    populationSize = globalCfg.populationSize;
    imWidth = globalCfg.targetImage.getWidth();
    imHeight = globalCfg.targetImage.getHeight();
    imSize = imWidth * imHeight;
    canvasSize = static_cast<i64>(imSize) * populationSize;
    if (canvasSize > std::numeric_limits<i32>::max()) {
        // FIXME: Implement support for 64-bits canvas
        std::fprintf(stderr, "CudaFitnessEngine: canvas size is too large\n");
        std::fprintf(stderr, "  This is a limitation of the current implementation, for now :)\n");
        std::abort();
    }

    triangles.reserve(populationSize * 100);
    fitnesses.resize(populationSize);

    hostIndividualInfo = new IndividualInfo[populationSize];

    deviceMalloc(&deviceFitnesses, populationSize);
    deviceMalloc(&deviceCanvas, canvasSize);
    deviceMalloc(&deviceImage, imSize);
    deviceMalloc(&deviceIndividualInfo, populationSize);

    Vec3f* hostImage = new Vec3f[imSize];
    defer { delete[] hostImage; };

    Color* target = reinterpret_cast<Color*>(globalCfg.targetImage.getData());
    i32 j = 0;

    for (i32 tileY = 0; tileY < (imHeight + 15)/16; ++tileY) {
        for (i32 tileX = 0; tileX < (imWidth + 15)/16; ++tileX) {
            for (i32 dy = 0; dy < 16; dy++) {
                for (i32 dx = 0; dx < 16; dx++) {
                    i32 y = tileY * 16 + dy;
                    i32 x = tileX * 16 + dx;
                    if (x >= imWidth || y >= imHeight)
                        continue;

                    i32 i = y * imWidth + x;
                    hostImage[j++] = (target[i].a / 255.0f) * fromColor(target[i]);
                }
            }
        }
    }
    if (j != imSize) {
        std::fprintf(stderr, "CudaFitnessEngine: j != imSize\n");
        std::abort();
    }

    copyHostToDevice(deviceImage, hostImage, imSize);

    // Limit of shared memory per block is 49152 bytes
    if (globalCfg.maxTriangles * sizeof(VecTriangle) > 49100) {
        std::fprintf(stderr, "CudaFitnessEngine: maxTriangles is too large\n");
        std::abort();
    }
}

CudaFitnessEngine::Engine::~Engine() {
    hipFree(deviceFitnesses);
    hipFree(deviceCanvas);
    hipFree(deviceImage);
    hipFree(deviceIndividualInfo);

    free(hostIndividualInfo);
}

inline static __device__
bool pointInTriangle(Vec2i p, Vec2i a, Vec2i b, Vec2i c) {
    auto sign = [](Vec2i p1, Vec2i p2, Vec2i p3) {
        return (p1.x - p3.x) * (p2.y - p3.y) - (p2.x - p3.x) * (p1.y - p3.y);
    };

    bool b1, b2, b3;
    b1 = sign(p, a, b) < 0;
    b2 = sign(p, b, c) < 0;
    b3 = sign(p, c, a) < 0;

    return ((b1 == b2) && (b2 == b3));
}

// inline static __device__
// Vec3f colorBlend(Vec3f dst, Color src) {
//     float alpha = src.a / 255.0;

//     return dst * (1.0f - alpha) + fromColor(src) * alpha;
// }

inline static __device__
Vec3f colorBlend(Vec3f dst, Vec3f src, f32 alpha) {
    return dst * (1.0f - alpha) + src * alpha;
}

inline static __device__
bool isTriangleInBounds(VecTriangle const& t, i32 x, i32 y) {
    i32 x2 = x + 16;
    i32 y2 = y + 16;

    i32 minX = min(t.a.x, min(t.b.x, t.c.x));
    i32 minY = min(t.a.y, min(t.b.y, t.c.y));
    i32 maxX = max(t.a.x, max(t.b.x, t.c.x));
    i32 maxY = max(t.a.y, max(t.b.y, t.c.y));

    bool bad = false;
    bad |= (minX >= x2);
    bad |= (maxX < x);
    bad |= (minY >= y2);
    bad |= (maxY < y);

    return !bad;
}

struct GPUImageInfo {
    Vec3f* canvas;
    i32 width, height;
    i32 size;
};

struct GPUDrawData {
    VecTriangle* triangles;
    IndividualInfo* info;
};

#if 1

static __global__
void drawTriangles(GPUImageInfo image, GPUDrawData data) {
    i32 width = image.width;
    i32 height = image.height;

    i32 tileX = blockIdx.x;
    i32 tileY = blockIdx.y;
    i32 i = blockIdx.z;

    VecTriangle* triangles;
    i32 numTriangles;

    triangles = data.triangles + data.info[i].offset;
    numTriangles = data.info[i].size;

    extern __shared__ OptimizedVecTriangle sharedTriangles[];
    __shared__ i32 numTrianglesShared;

    for (i32 j = threadIdx.x; j < numTriangles; j += blockDim.x) {
        VecTriangle const& triangle = triangles[j];
        sharedTriangles[j].a = triangle.a;
        sharedTriangles[j].b = triangle.b;
        sharedTriangles[j].c = triangle.c;
        sharedTriangles[j].color = fromColor(triangle.color);
        sharedTriangles[j].alpha = triangle.color.a / 255.0f;

        bool bounds = isTriangleInBounds(triangle, 16 * tileX, 16 * tileY);
        sharedTriangles[j].use = bounds;
    }

    constexpr i32 N = 32;

    __shared__ i32 mShared[N];
    __syncthreads();
    if (threadIdx.x < N) {
        i32 id = threadIdx.x;

        i32 ini = id * numTriangles / N;
        i32 fim = (id + 1) * numTriangles / N;
        fim = min(fim, numTriangles);
        OptimizedVecTriangle localTriangles[64];

        i32 m = 0;
        for (i32 j = ini; j < fim; j++) {
            OptimizedVecTriangle const& triangle = sharedTriangles[j];
            if (!triangle.use)
                continue;
            localTriangles[m++] = triangle;
        }
        mShared[id] = m;
        __syncwarp();
        if (threadIdx.x == 0) {
            for (i32 j = 1; j < N; j++)
                mShared[j] += mShared[j - 1];
            numTrianglesShared = mShared[N - 1];
        }
        __syncwarp();

        i32 m0 = id == 0 ? 0 : mShared[id - 1];
        for (i32 j = 0; j < m; j++) {
            sharedTriangles[m0 + j] = localTriangles[j];
        }
    }
    __syncthreads();

    i32 x = tileX * 16 + threadIdx.x % 16;
    i32 y = tileY * 16 + threadIdx.x / 16;

    if (x >= width || y >= height)
        return;

    Vec3f pixel = Vec3f{0.0, 0.0, 0.0};

    for (i32 j = 0; j < numTrianglesShared; ++j) {
        OptimizedVecTriangle const& t = sharedTriangles[j];

        if (pointInTriangle(Vec2i{x, y}, t.a, t.b, t.c)) {
            pixel = colorBlend(pixel, t.color, t.alpha);
        }
    }

    Vec3f* canvas = image.canvas + i * image.size;

    i32 m = 16;
    if (tileY == image.height / 16)
        m = image.height % 16;

    i32 xy = 16 * (tileY * width + m * tileX) + threadIdx.x;
    canvas[xy] = pixel;
}

#else

static __global__
void drawTriangles(GPUImageInfo image, GPUDrawData data) {
    i32 width = image.width;
    i32 height = image.height;

    i32 tileX = blockIdx.x;
    i32 tileY = blockIdx.y;
    i32 i = blockIdx.z;

    VecTriangle* triangles;
    i32 numTriangles;

    triangles = data.triangles + data.info[i].offset;
    numTriangles = data.info[i].size;

    extern __shared__ OptimizedVecTriangle sharedTriangles[];
    __shared__ i32 numTrianglesShared;

    for (i32 j = threadIdx.x; j < numTriangles; j += blockDim.x) {
        VecTriangle const& t = triangles[j];
        sharedTriangles[j].a = t.a;
        sharedTriangles[j].b = t.b;
        sharedTriangles[j].c = t.c;
        sharedTriangles[j].color = fromColor(t.color);
        sharedTriangles[j].alpha = t.color.a / 255.0f;

        bool bounds = isTriangleInBounds(t, 16 * tileX, 16 * tileY);
        sharedTriangles[j].use = bounds;
    }
    __syncthreads();
    if (threadIdx.x == 0) {
        i32 m = 0;
        for (i32 j = 0; j < numTriangles; j++) {
            if (!sharedTriangles[j].use)
                continue;

            i32 idx = m++;
            OptimizedVecTriangle const& triangle = sharedTriangles[j];
            sharedTriangles[idx] = triangle;
        }

        numTrianglesShared = m;
    }
    __syncthreads();

    i32 x = tileX * 16 + threadIdx.x % 16;
    i32 y = tileY * 16 + threadIdx.x / 16;

    if (x >= width || y >= height)
        return;

    Vec3f pixel = Vec3f{0.0, 0.0, 0.0};

    for (i32 j = 0; j < numTrianglesShared; ++j) {
        OptimizedVecTriangle const& t = sharedTriangles[j];

        if (pointInTriangle(Vec2i{x, y}, t.a, t.b, t.c)) {
            pixel = colorBlend(pixel, t.color, t.alpha);
        }
    }

    Vec3f* canvas = image.canvas + i * image.size;

    i32 m = 16;
    if (tileY == image.height / 16)
        m = image.height % 16;

    i32 xy = 16 * (tileY * width + m * tileX) + threadIdx.x;
    canvas[xy] = pixel;
}

#endif

static __global__
void computeFitnessKernel(
        Vec3f* target, Vec3f* canvas, 
        f64* fitness, i32 imWidth, i32 imHeight, i32 populationSize) {
    i32 numTasks = imWidth * imHeight;
    i32 threadId = blockDim.x * blockIdx.x + threadIdx.x;
    i32 i = threadId / THREADS_PER_INDIVIDUAL;
    if (i >= populationSize)
        return;
    i32 j = threadId % THREADS_PER_INDIVIDUAL;

    f64 fitnessSum = 0.0;
    i32 task0 = j * (numTasks / THREADS_PER_INDIVIDUAL) + min(j, numTasks % THREADS_PER_INDIVIDUAL);
    i32 n = numTasks / THREADS_PER_INDIVIDUAL + (j < numTasks % THREADS_PER_INDIVIDUAL);

    i32 imSize = imWidth * imHeight;
    for (i32 xy = task0; xy < task0 + n; ++xy) {
        Vec3f imgPixel = target[xy];
        Vec3f pixel = canvas[i * imSize + xy];

        double dx = imgPixel.x - pixel.x;
        double dy = imgPixel.y - pixel.y;
        double dz = imgPixel.z - pixel.z;

        fitnessSum += dx * dx + dy * dy + dz * dz;   
    }

    atomicAdd(&fitness[i], fitnessSum);
}

void CudaFitnessEngine::Engine::evaluate(std::vector<Individual>& individuals) {
    if (individuals.size() != populationSize) {
        std::fprintf(stderr, "CudaFitnessEngine::evaluate: individuals.size() != populationSize\n");
        std::abort();
    }

    defer { profiler.stop("cudaFitness:cleanup"); };

    // FIXME: Remove this
    // static i32 generation = 0;
    // i64 total = 0;
    // i64 amount_l = 0, amount_r = 0;

    profiler.start("cudaFitness:prepare", "Prepare");
    i32 maxTriangles = 0;
    triangles.clear();
    for (i32 i = 0; i < populationSize; ++i) {
        Individual const& ind = individuals[i];
        maxTriangles = std::max(maxTriangles, ind.size());

        hostIndividualInfo[i].offset = triangles.size();
        for (Triangle const& t : ind) {
            VecTriangle vt;
            vt.a = Vec2i(t.a.x, t.a.y);
            vt.b = Vec2i(t.b.x, t.b.y);
            vt.c = Vec2i(t.c.x, t.c.y);
            vt.color = t.color;

            triangles.push_back(vt);
        }

        hostIndividualInfo[i].size = ind.size();
    }
    profiler.stop("cudaFitness:prepare");

    // generation++;
    // total = triangles.size();
    // if (generation % 10 == 0) {
    //     double ratio = 100 * (amount_l + amount_r) / (double)total;
    //     std::fprintf(stderr, "Generation %d: %.2f%%\n", generation, ratio);
    // }

    profiler.start("cudaFitness:copy2device", "Copy");
    auto deviceTriangles = deviceMalloc<VecTriangle>(triangles.size());
    defer { hipFree(deviceTriangles); };
    copyHostToDevice(deviceTriangles, triangles.data(), triangles.size());
    copyHostToDevice(deviceIndividualInfo, hostIndividualInfo, populationSize);
    hipDeviceSynchronize();
    profiler.stop("cudaFitness:copy2device");

    profiler.start("cudaFitness:draw", "Draw");
    {
        // Must be 256 always
        constexpr u32 THREADS = 256;

        dim3 BLOCKS;
        BLOCKS.x = (imWidth + 15) / 16;
        BLOCKS.y = (imHeight + 15) / 16;
        BLOCKS.z = populationSize;

        GPUImageInfo imageInfo {
            .canvas = deviceCanvas,
            .width = imWidth,
            .height = imHeight,
            .size = imSize
        };

        GPUDrawData data {
            .triangles = deviceTriangles,
            .info = deviceIndividualInfo
        };

        drawTriangles<<<BLOCKS, THREADS,
            maxTriangles * sizeof(OptimizedVecTriangle)
        >>>(imageInfo, data);

        CUDA_CHECK(hipPeekAtLastError());
        CUDA_CHECK(hipDeviceSynchronize());
    }
    profiler.stop("cudaFitness:draw");

    profiler.start("cudaFitness:compute", "Compute");
    i32 N = populationSize * THREADS_PER_INDIVIDUAL;
    u32 THREADS = 128;
    u32 BLOCKS = (N + THREADS - 1) / THREADS;
    hipMemset(deviceFitnesses, 0, populationSize * sizeof(*deviceFitnesses));
    computeFitnessKernel<<<BLOCKS, THREADS>>>(
            deviceImage, deviceCanvas, deviceFitnesses, imWidth, imHeight, populationSize);
    CUDA_CHECK(hipPeekAtLastError());
    copyDeviceToHost(fitnesses.data(), deviceFitnesses, populationSize);
    hipDeviceSynchronize();
    profiler.stop("cudaFitness:compute");

    profiler.start("cudaFitness:copy2individuals", "Copy to individuals");
    for (i32 i = 0; i < populationSize; ++i) {
        individuals[i].setFitness(fitnesses[i]);
    }
    profiler.stop("cudaFitness:copy2individuals");


    profiler.start("cudaFitness:cleanup", "Cleanup");
}

// Wrapper for the actual implementation of the engine

CudaFitnessEngine::CudaFitnessEngine() :
    impl(std::make_unique<Engine>()) {}

CudaFitnessEngine::~CudaFitnessEngine() = default;

void CudaFitnessEngine::evaluate(std::vector<Individual>& individuals) {
    impl->evaluate(individuals);
}

GA_NAMESPACE_END

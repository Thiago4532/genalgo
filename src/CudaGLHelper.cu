#include "hip/hip_runtime.h"
#include "CudaGLHelper.hpp"

#include "Color.hpp"

#include <cstdio>
#include <cstdlib>
#include <cuda_gl_interop.h>
#include "Vec.hpp"
#include "GlobalConfig.hpp"
#include "defer.hpp"
#include "PoorProfiler.hpp"

#define CUDA_CHECK(expr) do { \
    hipError_t error = (expr); \
    if (error != hipSuccess) { \
        std::fprintf(stderr, "CUDA %s: %s\n", #expr, hipGetErrorString(error)); \
        std::abort(); \
    } \
} while (0)

GA_NAMESPACE_BEGIN

constexpr i32 THREADS_PER_INDIVIDUAL = 64;

static Vec3d fromColor(Color c) {
    return Vec3d{1.0 * c.r, 1.0 * c.g, 1.0 * c.b};
}

CudaGLHelper::CudaGLHelper() {
    width = globalCfg.targetImage.getWidth();
    height = globalCfg.targetImage.getHeight();
 
    Vec3d* imageInHost = new Vec3d[width * height];
    defer { delete[] imageInHost; };

    Color* target = reinterpret_cast<Color*>(globalCfg.targetImage.getData());
    for (i32 y = 0; y < height; ++y) {
        for (i32 x = 0; x < width; ++x) {
            i32 i = y * width + x;
            imageInHost[i] = (target[i].a / 255.0) * fromColor(target[i]);
        }
    }

    imageInDevice = nullptr;
    fitnessInDevice = nullptr;

    CUDA_CHECK(hipMalloc(&imageInDevice, width * height * sizeof(Vec3d)));
    CUDA_CHECK(hipMemcpy(imageInDevice, imageInHost, width * height * sizeof(Vec3d), hipMemcpyHostToDevice));

    CUDA_CHECK(hipMalloc(&fitnessInDevice, globalCfg.populationSize * sizeof(f64)));
}

CudaGLHelper::~CudaGLHelper() {
    hipFree(imageInDevice);
    hipFree(fitnessInDevice);
    unregisterTextures();
}

void CudaGLHelper::registerTextures(i32 count, u32 textures[]) {
    resources.resize(count);
    for (i32 i = 0; i < count; ++i)
        CUDA_CHECK(hipGraphicsGLRegisterImage(&resources[i], textures[i], GL_TEXTURE_2D, hipGraphicsRegisterFlagsReadOnly));
}

__global__ void computeFitnessKernel(
        Vec3d* target,
        hipTextureObject_t* textures,
        f64* fitness, i32 width, i32 height, i32 populationSize) {

    i32 numTasks = width * height;
    i32 threadId = blockDim.x * blockIdx.x + threadIdx.x;
    i32 i = threadId / THREADS_PER_INDIVIDUAL;
    if (i >= populationSize)
        return;
    i32 j = threadId % THREADS_PER_INDIVIDUAL;

    f64 fitnessSum = 0.0;
    i32 task0 = j * (numTasks / THREADS_PER_INDIVIDUAL) + min(j, numTasks % THREADS_PER_INDIVIDUAL);
    i32 n = numTasks / THREADS_PER_INDIVIDUAL + (j < numTasks % THREADS_PER_INDIVIDUAL);

    for (i32 xy = task0; xy < task0 + n; ++xy) {
        i32 x = xy % width;
        i32 y = xy / width;

        uchar4 color = tex2D<uchar4>(textures[i], x, y);
        double alpha = color.w / 255.0;

        Vec3d imgPixel = target[xy];
        double3 pixel = make_double3(alpha * color.x, alpha * color.y, alpha * color.z);

        double dx = imgPixel.x - pixel.x;
        double dy = imgPixel.y - pixel.y;
        double dz = imgPixel.z - pixel.z;

        fitnessSum += dx * dx + dy * dy + dz * dz;   
    }

    atomicAdd(&fitness[i], fitnessSum);
}

void CudaGLHelper::computeFitness(std::vector<f64>& fitness) {
    if (fitness.size() != resources.size()) {
        std::fprintf(stderr, "CudaGLHelper::computeFitness: fitness.size() != resources.size()\n");
        std::abort();
    }

    defer { profiler.stop("Cleanup"); };

    profiler.start("Mapping");
    std::vector<hipArray_t> textureArrays(resources.size());
    std::vector<hipTextureObject_t> textureObjects(resources.size());

    CUDA_CHECK(hipGraphicsMapResources(resources.size(), resources.data()));
    defer { hipGraphicsUnmapResources(resources.size(), resources.data()); };
    profiler.stop("Mapping");
    profiler.start("Textures");

    hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeArray;

    hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.addressMode[0] = hipAddressModeClamp;
    texDesc.addressMode[1] = hipAddressModeClamp;
    texDesc.filterMode = hipFilterModePoint;
    texDesc.readMode = hipReadModeElementType;
    texDesc.normalizedCoords = 0;

    for (i32 i = 0; i < resources.size(); ++i) {
        // Get mapped arrays
        CUDA_CHECK(hipGraphicsSubResourceGetMappedArray(&textureArrays[i], resources[i], 0, 0));
        
        // Update texture object
        resDesc.res.array.array = textureArrays[i];
        CUDA_CHECK(hipCreateTextureObject(&textureObjects[i], &resDesc, &texDesc, nullptr));
    }
    profiler.stop("Textures");

    defer {
        profiler.start("Cleanup");
        for (i32 i = 0; i < resources.size(); ++i)
            hipDestroyTextureObject(textureObjects[i]);
        profiler.stop("Cleanup");
    };

    profiler.start("Copy");
    hipTextureObject_t* d_textureObjects;
    CUDA_CHECK(hipMalloc(&d_textureObjects, textureObjects.size() * sizeof(hipTextureObject_t)));
    defer { hipFree(d_textureObjects); };
    CUDA_CHECK(hipMemcpy(d_textureObjects, textureObjects.data(), textureObjects.size() * sizeof(hipTextureObject_t), hipMemcpyHostToDevice));
    profiler.stop("Copy");

    profiler.start("Kernel");

    // Launch kernel
    i32 N = resources.size() * THREADS_PER_INDIVIDUAL;
    i32 THREADS = 128;
    i32 BLOCKS = (N + THREADS - 1) / THREADS;

    hipMemset(fitnessInDevice, 0, fitness.size() * sizeof(f64));
    computeFitnessKernel<<<BLOCKS, THREADS>>>(
            imageInDevice,
            d_textureObjects,
            fitnessInDevice, width, height, globalCfg.populationSize);
    CUDA_CHECK(hipMemcpy(fitness.data(), fitnessInDevice, fitness.size() * sizeof(f64), hipMemcpyDeviceToHost));
    profiler.stop("Kernel");

    profiler.start("Cleanup");
}

void CudaGLHelper::unregisterTextures() {
    // TODO
}

GA_NAMESPACE_END

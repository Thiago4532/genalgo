#include "hip/hip_runtime.h"
#include "Color.hpp"
#include "CudaGLHelper.hpp"

#include <cstdio>
#include <cstdlib>
#include <cuda_gl_interop.h>
#include "Vec.hpp"
#include "globalConfig.hpp"
#include "defer.hpp"

#define CUDA_CHECK(expr) do { \
    hipError_t error = (expr); \
    if (error != hipSuccess) { \
        std::fprintf(stderr, "CUDA %s: %s\n", #expr, hipGetErrorString(error)); \
        std::abort(); \
    } \
} while (0)

GA_NAMESPACE_BEGIN

static Vec3d fromColor(Color c) {
    return Vec3d{1.0 * c.r, 1.0 * c.g, 1.0 * c.b};
}

CudaGLHelper::CudaGLHelper() {
    width = globalCfg.targetImage.getWidth();
    height = globalCfg.targetImage.getHeight();
 
    Vec3d* imageInHost = new Vec3d[width * height];
    defer { delete[] imageInHost; };

    Color* target = reinterpret_cast<Color*>(globalCfg.targetImage.getData());
    for (i32 y = 0; y < height; ++y) {
        for (i32 x = 0; x < width; ++x) {
            i32 i = y * width + x;
            imageInHost[i] = (target[i].a / 255.0) * fromColor(target[i]);
        }
    }

    imageInDevice = nullptr;
    fitnessInDevice = nullptr;

    CUDA_CHECK(hipMalloc(&imageInDevice, width * height * sizeof(Vec3d)));
    CUDA_CHECK(hipMemcpy(imageInDevice, imageInHost, width * height * sizeof(Vec3d), hipMemcpyHostToDevice));

    CUDA_CHECK(hipMalloc(&fitnessInDevice, globalCfg.populationSize * sizeof(f64)));
}

CudaGLHelper::~CudaGLHelper() {
    hipFree(imageInDevice);
    hipFree(fitnessInDevice);
    unregisterTextures();
}

void CudaGLHelper::registerTextures(i32 count, u32 textures[]) {
    resources.resize(count);
    for (i32 i = 0; i < count; ++i)
        CUDA_CHECK(hipGraphicsGLRegisterImage(&resources[i], textures[i], GL_TEXTURE_2D, hipGraphicsRegisterFlagsReadOnly));
}

__global__ void computeFitnessKernel(
        Vec3d* target,
        hipTextureObject_t* textures,
        f64* fitness, i32 width, i32 height) {

    i32 i = threadIdx.x;
    f64 fitnessSum = 0.0;

    for (i32 y = 0; y < height; ++y) {
        for (i32 x = 0; x < width; ++x) {
            uchar4 color = tex2D<uchar4>(textures[i], x, y);
            double alpha = color.w / 255.0;

            Vec3d imgPixel = target[y * width + x];
            double3 pixel = make_double3(alpha * color.x, alpha * color.y, alpha * color.z);

            double dx = imgPixel.x - pixel.x;
            double dy = imgPixel.y - pixel.y;
            double dz = imgPixel.z - pixel.z;
            
            fitnessSum += dx * dx + dy * dy + dz * dz;   
        }
    }

    fitness[i] = fitnessSum;
}

void CudaGLHelper::computeFitness(std::vector<f64>& fitness) {
    if (fitness.size() != resources.size()) {
        std::fprintf(stderr, "CudaGLHelper::computeFitness: fitness.size() != resources.size()\n");
        std::abort();
    }

    std::vector<hipArray_t> textureArrays(resources.size());
    std::vector<hipTextureObject_t> textureObjects(resources.size());

    CUDA_CHECK(hipGraphicsMapResources(resources.size(), resources.data()));
    defer { hipGraphicsUnmapResources(resources.size(), resources.data()); };

    hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeArray;

    hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.addressMode[0] = hipAddressModeClamp;
    texDesc.addressMode[1] = hipAddressModeClamp;
    texDesc.filterMode = hipFilterModePoint;
    texDesc.readMode = hipReadModeElementType;
    texDesc.normalizedCoords = 0;

    for (i32 i = 0; i < resources.size(); ++i) {
        // Get mapped arrays
        CUDA_CHECK(hipGraphicsSubResourceGetMappedArray(&textureArrays[i], resources[i], 0, 0));
        
        // Update texture object
        resDesc.res.array.array = textureArrays[i];
        CUDA_CHECK(hipCreateTextureObject(&textureObjects[i], &resDesc, &texDesc, nullptr));
    }

    defer {
        for (i32 i = 0; i < resources.size(); ++i)
            hipDestroyTextureObject(textureObjects[i]);
    };

    hipTextureObject_t* d_textureObjects;
    CUDA_CHECK(hipMalloc(&d_textureObjects, textureObjects.size() * sizeof(hipTextureObject_t)));
    defer { hipFree(d_textureObjects); };
    CUDA_CHECK(hipMemcpy(d_textureObjects, textureObjects.data(), textureObjects.size() * sizeof(hipTextureObject_t), hipMemcpyHostToDevice));

    // Launch kernel
    i32 BLOCKS = 1;
    i32 THREADS = resources.size();
    computeFitnessKernel<<<BLOCKS, THREADS>>>(
            imageInDevice,
            d_textureObjects,
            fitnessInDevice, width, height);
    CUDA_CHECK(hipMemcpy(fitness.data(), fitnessInDevice, fitness.size() * sizeof(f64), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipDeviceSynchronize());
}

void CudaGLHelper::unregisterTextures() {
    // TODO
}

GA_NAMESPACE_END
